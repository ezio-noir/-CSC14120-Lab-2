
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call)\
{\
	const hipError_t error = call;\
	if (error != hipSuccess)\
	{\
		fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
		fprintf(stderr, "code: %d, reason: %s\n", error,\
				hipGetErrorString(error));\
		exit(EXIT_FAILURE);\
	}\
}
#define TILE_WIDTH 32
struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

__global__ void matrix_multiplication_kernel1(float* A, float* B, float* C, int m, int n, int k)
{
	int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    float C_rc = 0;
    if (r < m && c < k) {
        for (int i = 0; i < n; ++i) {
            C_rc += A[r * n + i] * B[i * k + c];
        }
    }
    C[r * k + c] = C_rc;
}

__global__ void matrix_multiplication_kernel2(float* A, float* B, float* C, int m, int n, int k)
{
	__shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float s_B[TILE_WIDTH][TILE_WIDTH];

    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    float C_rc = 0;

    int r_A, c_A, r_B, c_B;
    for (int tile = 0; tile < (n - 1) / TILE_WIDTH + 1; ++tile) {
        r_A = r; c_A = tile * TILE_WIDTH + threadIdx.x;
        r_B = tile * TILE_WIDTH + threadIdx.y; c_B = c;
        s_A[threadIdx.y][threadIdx.x] = (r_A < m && c_A < n) ? A[r_A * n + c_A] : 0;
        s_B[threadIdx.y][threadIdx.x] = (r_B < n && c_B < k) ? B[r_B * k + c_B] : 0;
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i) {
            C_rc += s_A[threadIdx.y][i] * s_B[i][threadIdx.x];
        }
        __syncthreads();
    }

    C[r * k + c] = C_rc;
}

void matrix_multiplication(float* A, float* B, float* C, int m, int n, int k,
    bool useDevice = false, dim3 blockSize = dim3(1),int kernelType=1)
{
    GpuTimer timer;
    timer.Start();
    if (useDevice == false)
    {
        for (int r = 0; r < m; ++r) {
            for (int c = 0; c < k; ++c) {
                C[r * k + c] = 0;
                for (int i = 0; i < n; ++i) {
                    C[r * k + c] += A[r * n + i] * B[i * k + c];
                }
            }
        }
    }
    else // Use device
    {
        // TODO: Allocate device memories
        float* d_A, * d_B, * d_C;
        CHECK(hipMalloc((void**) &d_A, m * n * sizeof(float)));
        CHECK(hipMalloc((void**) &d_B, n * k * sizeof(float)));
        CHECK(hipMalloc((void**) &d_C, m * k * sizeof(float)));

        // TODO: Copy data to device memories
        CHECK(hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice));
        
        dim3 gridSize((m - 1) / blockSize.x + 1, (k - 1) / blockSize.y + 1); // TODO: Compute gridSize
        
		if (kernelType == 1)
			matrix_multiplication_kernel1<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);
		else if (kernelType == 2)
			matrix_multiplication_kernel2<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);

        // TODO: Copy result from device memory
        CHECK(hipMemcpy(C, d_C, m * k * sizeof(float), hipMemcpyDeviceToHost));

        // TODO: Free device memories
        CHECK(hipFree(d_A));
        CHECK(hipFree(d_B));
        CHECK(hipFree(d_C));
		
		printf("Grid size: %d * %d, block size: %d * %d\n", 
			gridSize.x,gridSize.y, blockSize.x,blockSize.y);

    }
    timer.Stop();
    float time = timer.Elapsed();
    printf("Processing time (%s): %f ms\n",
        useDevice == true ? "use device" : "use host", time);
}

float checkCorrectness(float * a1, float* a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)	
		err += abs(a1[i] - a2[i]);
	err /= n;
	return err;
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("****************************\n\n");

}
int main(int argc, char** argv)
{
	printDeviceInfo();
	
	//Declare variables
    float* h_A; // The A matrix
    float* h_B; // The B matrix
    float* h_C; // The output C matrix
    float* correct_C; // The output C matrix

    int m;    // number of rows in the matrix A
    int n; // number of columns in the matrix A, number of rows in the matrix B
    int k; // number of columns in the matrix B

    m = (1 << 10);
    n = (1 << 9);
    k = (1 << 10);

    // Set up input data
    h_A = (float*)malloc(m * n * sizeof(float));
    h_B = (float*)malloc(n * k * sizeof(float));
    h_C = (float*)malloc(m * k * sizeof(float));
    correct_C = (float*)malloc(m * k * sizeof(float));

    for (int i = 0; i < m; i++)
        for (int j = 0;j < n;j++)
            h_A[i*n+j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
 
    for (int i = 0; i < n; i++)
        for (int j = 0;j < k;j++)
            h_B[i*k+j] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);


    // Add vectors (on host)
    matrix_multiplication(h_A,h_B,correct_C,m,n,k);
	printf("\n");

	dim3 blockSize(32, 32); // Default
	if (argc == 3)
	{
		blockSize.x = atoi(argv[1]);
		blockSize.y = atoi(argv[2]);
	} 
    // Add in1 & in2 on device
	printf("Basic Matrix Multiplication:\n");
    matrix_multiplication(h_A, h_B, h_C, m, n, k, true,blockSize,1);
	float err = checkCorrectness(h_C, correct_C,m*k);
	printf("Error between device result and host result: %f\n\n", err);

	printf("Shared memory Matrix Multiplication:\n");
    matrix_multiplication(h_A, h_B, h_C, m, n, k, true,blockSize,2);
	err = checkCorrectness(h_C, correct_C,m*k);
	printf("Error between device result and host result: %f", err);	
	
    free(h_A);
    free(h_B);
    free(h_C);
    free(correct_C);

    return 0;
}
